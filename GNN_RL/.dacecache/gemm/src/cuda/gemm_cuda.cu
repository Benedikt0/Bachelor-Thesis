
#include <hip/hip_runtime.h>
#include <dace/dace.h>



DACE_EXPORTED int __dace_init_cuda(float * __restrict__ A, float * __restrict__ B, float * __restrict__ Y);
DACE_EXPORTED void __dace_exit_cuda(float * __restrict__ A, float * __restrict__ B, float * __restrict__ Y);



namespace dace { namespace cuda {
    hipStream_t __streams[2];
    hipEvent_t __events[2];
    int num_streams = 2;
    int num_events = 2;
} }

int __dace_init_cuda(float * __restrict__ A, float * __restrict__ B, float * __restrict__ Y) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    hipMalloc((void **) &dev_X, 1);
    hipFree(dev_X);

    // Create cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamCreateWithFlags(&dace::cuda::__streams[i], hipStreamNonBlocking);
    }
    for(int i = 0; i < 2; ++i) {
        hipEventCreateWithFlags(&dace::cuda::__events[i], hipEventDisableTiming);
    }

    

    return 0;
}

void __dace_exit_cuda(float * __restrict__ A, float * __restrict__ B, float * __restrict__ Y) {
    

    // Destroy cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamDestroy(dace::cuda::__streams[i]);
    }
    for(int i = 0; i < 2; ++i) {
        hipEventDestroy(dace::cuda::__events[i]);
    }
}


