
#include <hip/hip_runtime.h>
#include <dace/dace.h>



DACE_EXPORTED int __dace_init_cuda(float * __restrict__ result, float * __restrict__ x, float * __restrict__ y, int n);
DACE_EXPORTED void __dace_exit_cuda(float * __restrict__ result, float * __restrict__ x, float * __restrict__ y, int n);



namespace dace { namespace cuda {
    hipStream_t __streams[2];
    hipEvent_t __events[1];
    int num_streams = 2;
    int num_events = 1;
} }

int __dace_init_cuda(float * __restrict__ result, float * __restrict__ x, float * __restrict__ y, int n) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    hipMalloc((void **) &dev_X, 1);
    hipFree(dev_X);

    // Create cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamCreateWithFlags(&dace::cuda::__streams[i], hipStreamNonBlocking);
    }
    for(int i = 0; i < 1; ++i) {
        hipEventCreateWithFlags(&dace::cuda::__events[i], hipEventDisableTiming);
    }

    

    return 0;
}

void __dace_exit_cuda(float * __restrict__ result, float * __restrict__ x, float * __restrict__ y, int n) {
    

    // Destroy cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamDestroy(dace::cuda::__streams[i]);
    }
    for(int i = 0; i < 1; ++i) {
        hipEventDestroy(dace::cuda::__events[i]);
    }
}


